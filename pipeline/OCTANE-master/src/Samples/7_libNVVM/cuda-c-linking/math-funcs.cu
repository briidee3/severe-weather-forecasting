
#include <hip/hip_runtime.h>
// Copyright (c) 1993-2023, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

// Simple implementation of Mandelbrot set from Wikipedia
// http://en.wikipedia.org/wiki/Mandelbrot_set

// Note that this kernel is meant to be a simple, straight-forward
// implementation, and so may not represent optimized GPU code.
extern "C"
__device__
void mandelbrot(float* Data) {

  // Which pixel am I?
  unsigned DataX = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned DataY = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned Width = gridDim.x * blockDim.x;
  unsigned Height = gridDim.y * blockDim.y;

  float R, G, B, A;

  // Scale coordinates to (-2.5, 1) and (-1, 1)

  float NormX = (float)DataX / (float)Width;
  NormX *= 3.5f;
  NormX -= 2.5f;

  float NormY = (float)DataY / (float)Height;
  NormY *= 2.0f;
  NormY -= 1.0f;

  float X0 = NormX;
  float Y0 = NormY;

  float X = 0.0f;
  float Y = 0.0f;

  unsigned Iter = 0;
  unsigned MaxIter = 1000;

  // Iterate
  while(X*X + Y*Y < 4.0f && Iter < MaxIter) {
    float XTemp = X*X - Y*Y + X0;
    Y = 2.0f*X*Y + Y0;

    X = XTemp;

    Iter++;
  }

  unsigned ColorG = Iter % 50;
  unsigned ColorB = Iter % 25;

  R = 0.0f;
  G = (float)ColorG / 50.0f;
  B = (float)ColorB / 25.0f;
  A = 1.0f;

  Data[DataY*Width*4+DataX*4+0] = R;
  Data[DataY*Width*4+DataX*4+1] = G;
  Data[DataY*Width*4+DataX*4+2] = B;
  Data[DataY*Width*4+DataX*4+3] = A;
}
